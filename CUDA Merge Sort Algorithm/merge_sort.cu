#include "hip/hip_runtime.h"
#include "merge_sort.h"

#define min(a, b) (a < b ? a : b)
// Based on https://github.com/kevin-albert/cuda-mergesort/blob/master/mergesort.cu

__host__ std::tuple<dim3, dim3, int> parseCommandLineArguments(int argc, char** argv) 
{
    int numElements = 32;
    dim3 threadsPerBlock;
    dim3 blocksPerGrid;

    threadsPerBlock.x = 32;
    threadsPerBlock.y = 1;
    threadsPerBlock.z = 1;

    blocksPerGrid.x = 8;
    blocksPerGrid.y = 1;
    blocksPerGrid.z = 1;

    for (int i = 1; i < argc; i++) {
        if (argv[i][0] == '-' && argv[i][1] && !argv[i][2]) {
            char arg = argv[i][1];
            unsigned int* toSet = 0;
            switch(arg) {
                case 'x':
                    toSet = &threadsPerBlock.x;
                    break;
                case 'y':
                    toSet = &threadsPerBlock.y;
                    break;
                case 'z':
                    toSet = &threadsPerBlock.z;
                    break;
                case 'X':
                    toSet = &blocksPerGrid.x;
                    break;
                case 'Y':
                    toSet = &blocksPerGrid.y;
                    break;
                case 'Z':
                    toSet = &blocksPerGrid.z;
                    break;
                case 'n':
                    i++;
                    numElements = std::stoi(argv[i]);
                    break;
            }
            if (toSet) {
                i++;
                *toSet = (unsigned int) strtol(argv[i], 0, 10);
            }
        }
    }
    return {threadsPerBlock, blocksPerGrid, numElements};
}

__host__ long* generateRandomLongArray(int numElements)
{
    //TODO generate random array of long integers of size numElements
    long *randomLongs;
    randomLongs = (long*) malloc(sizeof(long)*numElements);
    if (randomLongs == NULL) {
        printf("Memory allocation failed\n");
        return NULL; // Return NULL if malloc failed
    }

    for(int i=0;i<numElements;i++){
        randomLongs[i] = rand()%1000;
    }

    return randomLongs;
}

__host__ void printHostMemory(long *host_mem, int num_elments)
{
    // Output results
    for(int i = 0; i < num_elments; i++)
    {
        printf("%ld ",host_mem[i]);
    }
    printf("\n");
}

__host__ int main(int argc, char** argv) 
{

    auto[threadsPerBlock, blocksPerGrid, numElements] = parseCommandLineArguments(argc, argv);

    long *data = generateRandomLongArray(numElements);

    printf("Unsorted data: ");
    printHostMemory(data, numElements);

    data = mergesort(data, numElements, threadsPerBlock, blocksPerGrid);

    printf("Sorted data: ");
    printHostMemory(data, numElements);
}

__host__ std::tuple <long* ,long* ,dim3* ,dim3*> allocateMemory(int numElements)
{
    //
    // Allocate two arrays on the GPU
    // we switch back and forth between them during the sort
    //
    long* D_data;
    long* D_swp;
    dim3* D_threads;
    dim3* D_blocks;

    // Actually allocate the two arrays
    hipMalloc((long**) &D_data, numElements*sizeof(long));
    hipMalloc((long**) &D_swp, numElements*sizeof(long));

    hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("CUDA error: %s\n", hipGetErrorString(error));
		// system("pause"); // when using VisStudio
		exit(-1);
	}

    // Copy the thread / block info to the GPU as well
    hipMalloc((long**) &D_threads, numElements*sizeof(dim3));
    hipMalloc((long**) &D_blocks, numElements*sizeof(dim3));

    return {D_data, D_swp, D_threads, D_blocks};
}

__host__ long* mergesort(long* data, long numElements, dim3 threadsPerBlock, dim3 blocksPerGrid) {

    auto[D_data, D_swp, D_threads, D_blocks] = allocateMemory(numElements);

    // Copy from our input list into the first array
    hipMemcpy(D_data, data, numElements * sizeof(long), hipMemcpyHostToDevice);
    hipMemcpy(D_threads, &threadsPerBlock, sizeof(dim3), hipMemcpyHostToDevice);
    hipMemcpy(D_threads, &blocksPerGrid, sizeof(dim3), hipMemcpyHostToDevice);

    long* A = D_data;
    long* B = D_swp;

    long nThreads = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z *
                    blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z;

    // TODO Initialize timing metrics variable(s). The implementation of this is up to you
    
    //
    // Slice up the list and give pieces of it to each thread, letting the pieces grow
    // bigger and bigger until the whole list is sorted
    //
    for (int width = 2; width < (numElements << 1); width <<= 1) {
        long slices = numElements / ((nThreads) * width) + 1;

        // Actually call the kernel
        gpu_mergesort<<<blocksPerGrid, threadsPerBlock>>>(A, B, numElements, width, slices, D_threads, D_blocks); //TODO You will need to populate arguments for the kernel

        // Switch the input / output arrays instead of copying them around
        A = A==D_data?D_swp:D_data;
        B = B==D_data?D_swp:D_data;
    }

    // Get the list back from the GPU
    hipMemcpy(data, A, numElements * sizeof(long), hipMemcpyDeviceToHost);

    // TODO calculate and print to stdout kernel execution time

    // Free the GPU memory
    hipFree(A);
    hipFree(B);
    return data;
}

// GPU helper function
// calculate the id of the current thread
__device__ unsigned int getIdx(dim3* threads, dim3* blocks) {
    int x;
    return threadIdx.x +
           threadIdx.y * (x  = threads->x) +
           threadIdx.z * (x *= threads->y) +
           blockIdx.x  * (x *= threads->z) +
           blockIdx.y  * (x *= blocks->z) +
           blockIdx.z  * (x *= blocks->y);
}

//
// Perform a full mergesort on our section of the data.
//
__global__ void gpu_mergesort(long* source, long* dest, long size, long width, long slices, dim3* threads, dim3* blocks) {
    unsigned int idx = getIdx(threads, blocks);
    // TODO initialize 3 long variables start, middle, and end
    // middle and end do not have values set,
    // while start is set to the width of the merge sort data span * the thread index * number of slices that this kernel will sort
    long start=width*idx*slices, 
    middle, 
    end;

    for (long slice = 0; slice < slices; slice++) {
        // Break from loop when the start variable is >= size of the input array
        if(start>=size){
            break;
        }

        // Set middle to be minimum middle index (start index plus 1/2 width) and the size of the input array
        middle = min(start + (width>>1), size);

        // Set end to the minimum of the end index (start index plus the width of the current data window) and the size of the input array
        end = min(start+width, size);

        // Perform bottom up merege given the two available arrays and the start, middle, and end variables
        gpu_bottomUpMerge(source, dest, start, middle, end);

        // Increase the start index by the width of the current data window
        start += width;
    }
}

//
// Finally, sort something gets called by gpu_mergesort() for each slice
// Note that the pseudocode below is not necessarily 100% complete you may want to review the merge sort algorithm.
//
__device__ void gpu_bottomUpMerge(long* source, long* dest, long start, long middle, long end) {
    long i = start;
    long j = middle;

    // Create a for look that iterates between the start and end indexes
    for (long k=start;k<end;k++) {
        // if i is before the middle index and (j is the final index or the value at i <  the value at j)
        if (i<middle && (j>=end || source[i]<source[j])) {
            // set the value in the destination array at index k to the value at index i in the source array
            dest[k] = source[i];
            // increment i
            i++;
        } else {
            // set the value in the destination array at index k to the value at index j in the source array
            dest[k] = source[j];
            // increment j
            j++;
        }
    }
}